
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

// GPU Code
// __global__ indicates that it is a GPU kernel, that can be called from the CPU
__global__ void Add(float* d_a, float* d_b, float* d_c, int N)
{
         int id = blockIdx.x * blockDim.x + threadIdx.x;
         if(id < N)
           
            d_c[id] = d_a[id] + d_b[id];
    
}

// CPU Code
int main()
{   
    int N;
    cout << "Enter the array size : ";
    cin >> N;
    float h_a[N], h_b[N], h_c[N];

    int Array_Bytes = N * sizeof(float);  

    for(int i=0; i<N; i++)
    {
        h_a[i] = i;
    }
    for(int i=0;i<N;i++)
    {
    h_b[i]=i+1;
    }
 

    // Declaring pointers for allocation on the device 
    float* d_a;
    float* d_b;
    float* d_c;

    // Allocating device memory
    hipMalloc((void**)&d_a,  Array_Bytes);
    hipMalloc((void**)&d_b,  Array_Bytes);
    hipMalloc((void**)&d_c,  Array_Bytes);

    // Copying input operands from host to device
    // For the GPU to perform any operation, the data has to be present in the GPU memory
    hipMemcpy(d_a, h_a,  Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b,  Array_Bytes, hipMemcpyHostToDevice);

    // Launching kernel with 1 block and 1 thread
    // The host launches the kernel on the device
   
    Add<<<ceil(1.0*N/1024), 1024>>>(d_a, d_b, d_c,N);

    // Copying the result from device to host
    hipMemcpy(h_c, d_c, Array_Bytes, hipMemcpyDeviceToHost);
     

   int flag=0;
    
        
    cout<<"\n";
    for(int i=0;i<N;i++)
    {
    if(h_a[i]+h_b[i]!=h_c[i])
    flag=1;
    }
    
    if(flag)
    cout<<"result error";
    else
    cout<<"correct result";
    
    hipFree(d_a);    
    hipFree(d_b);       
    hipFree(d_c);
}
